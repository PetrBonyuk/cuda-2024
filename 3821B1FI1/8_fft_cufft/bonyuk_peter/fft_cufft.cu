/*When entering the following vector:
int batch = 1;
    std::vector<float> input = {1.0, 0.0, 2.0, 0.0, 3.0, 0.0, 4.0, 0.0}; 
The output values ​​were:
1 0 2 0 3 0 4 0 
*/


#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <iostream>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);

    hipfftComplex* dIn;
    hipfftComplex* dOut;
    std::vector<float> output(2 * n * batch);

    hipMalloc((void**)&dIn, sizeof(hipfftComplex) * n * batch);
    hipMalloc((void**)&dOut, sizeof(hipfftComplex) * n * batch);

    hipMemcpy(dIn, input.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, dIn, dOut, HIPFFT_FORWARD);

    hipfftExecC2C(plan, dOut, dIn, HIPFFT_BACKWARD);

    float normalizationFactor = 1.0f / n;
    hipMemcpy(output.data(), dIn, sizeof(float) * output.size(), hipMemcpyDeviceToHost);
    for (int i = 0; i < output.size(); ++i) {
        output[i] *= normalizationFactor;
    }

    hipfftDestroy(plan);
    hipFree(dIn);
    hipFree(dOut);

    return output;
}